#include "hip/hip_runtime.h"
#ifndef IPSP3D
#define IPSP3D
#define cudaCheck(input){cudaAssert((input), __FILE__, __LINE__); } // http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api

#include <iostream>
#include <stdlib.h>
#include <cstring>
#include <vector>
#include <hipblas.h>
#include <stdio.h>

inline void cudaAssert(hipError_t code, const char *file, int line){
	if (code != hipSuccess){ fprintf(stderr, "CudaAssert %s %s %d\n", hipGetErrorString(code), file, line); exit(code);}
}

void printDeviceDetails(){
    std::cout << "Device Details... \n" << std::endl;
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        int driverVersion = 0, runtimeVersion = 0;
        hipRuntimeGetVersion(&runtimeVersion);
        hipDriverGetVersion(&driverVersion); // http://rcs.bu.edu/examples/gpu/tutorials/deviceQuery/deviceQuery.cpp
        printf("  Driver Version: %d.%d\n",driverVersion/1000, (driverVersion%100)/10);
        printf("  Runtime Version: %d.%d\n",runtimeVersion/1000, (runtimeVersion%100)/10);
        printf("  CUDA Version: %d.%d\n\n",  prop.major, prop.minor);
    }
}

class h_Matrix{
public:
    double* elements;
    double* devElements;
    int height, width, depth;
    __host__ __device__ h_Matrix() : height(1), width(1), depth(1) {};
    __host__ __device__ h_Matrix(int height, int width, int depth) : height(height), width(width), depth(depth) { elements = new double[height*width*depth](); };
    __host__ __device__ h_Matrix(double* elements, int height, int width, int depth) : height(height), width(width), depth(depth), elements(elements) {};
    __host__ __device__ int numel (){return height * width * depth;};
    __host__ __device__ double* getColDouble(int i){return &elements[height * i];}
    __host__ __device__ double* getElement(int i, int j){return &elements[i * height + j];};
    __host__ __device__ void setElement(int i, int j, double value){elements[i * height + j] = value;};
    __host__ __device__ void setElement(int i, double value){elements[i] = value;};
    __host__ __device__ double* getElement(int i){return &elements[i];};
    __host__ __device__ h_Matrix getCol(int i){h_Matrix newMatrix(getColDouble(i),height, 1, 1); return newMatrix;};
    __host__ __device__ h_Matrix getPlane(int i){h_Matrix newMatrix(&elements[height * width * i], height, width, 1); return newMatrix;};
};



__device__ void multiplyCuda(double* a, double* b, double* c, int lda, int ldb, int ldc, int m, int n, int k, hipblasOperation_t op1, hipblasOperation_t op2, double* alpha, double* beta){
	// m is number of rows of op(a)
	// n is number of cols of op(b)
	// k is number of rows of a and width of c)

	int y = threadIdx.y; //col
	int x = threadIdx.x; //row

	if (y < m && x < n){
	   double cellSum = 0;
			 if (op1 == HIPBLAS_OP_N && op2 == HIPBLAS_OP_N){
				 for(int i = 0; i < k; i++){
					cellSum += a[lda * i + y] * b[ldb * x + i] * alpha[0];
					//printf("\ni: %d, y: %d, x: %d, lda: %d, ldb: %d, alpha: %f, temp: %f, cellSum: %f, aVal: %f, bVal: %f, aind %d, bind: %d", i, y, x, lda, ldb, alpha[0], a[lda * i + y] * b[ldb * x + i] * alpha[0], cellSum, a[lda * i + y], b[ldb * x + i], lda*i+y, ldb*x+i);
				 }
			 }else if(op1 == HIPBLAS_OP_T && op2 == HIPBLAS_OP_N){
				 for(int i = 0; i < k; i++){
					cellSum += a[lda * y + i] * b[ldb * x + i] * alpha[0];
				 }
			 }else if(op1 == HIPBLAS_OP_N && op2 == HIPBLAS_OP_T){
				 for(int i = 0; i < k; i++){
					cellSum += a[lda * i + y] * b[ldb * i + x] * alpha[0];
				 }
			 }else if(op1 == HIPBLAS_OP_T && op2 == HIPBLAS_OP_T){
				 for(int i = 0; i < k; i++){
					cellSum += a[lda * y + i] * b[ldb * i + x] * alpha[0];
				 }
			 }
			c[ldc * y + x] = beta[0] * c[ldc * y + x] + cellSum;
			//printf("\nThreadID %d,%d, A: %f, / %d    B: %f, / %d    C: %f / %d \t K: %d, M: %d, N: %d\n", x,y,a[lda * x + y],lda * x + y,b[ldb * y + x],ldb * y + x,c[ldc * y + x],ldc * y + x, k, m, n);
   }
}

__device__ void matrixMultiplyCuda(h_Matrix* a, h_Matrix* b, h_Matrix* c, int m, int n, int k, hipblasOperation_t op1, hipblasOperation_t op2, double* alpha, double* beta){
	int lda = a->height;
	int ldb = b->height;
	int ldc = m;
	multiplyCuda(a->elements, b->elements, c->elements,lda, ldb, ldc, m, n, k, op1, op2, alpha, beta);
	c->height = ldc;
	c->width = k;
}



__device__ void matrixMultiplyCuda(h_Matrix* a, h_Matrix* b, double* c, int m, int n, int k, hipblasOperation_t op1, hipblasOperation_t op2, double* alpha, double* beta){
	int lda = a->height;
	int ldb = b->height;
	int ldc = m;
	multiplyCuda(a->elements, b->elements, c,lda, ldb, ldc, m, n, k, op1, op2, alpha, beta);
}

__device__ void matrixMultiplyCuda(h_Matrix* a, h_Matrix* b, h_Matrix* c, int m, int n, int k, hipblasOperation_t op1, hipblasOperation_t op2, double* alpha){
	double* beta = new double[1]();
	matrixMultiplyCuda(a, b, c, m, n, k, op1, op2, alpha, beta);
}




__device__ void matrixMultiplyCuda(h_Matrix* a, h_Matrix* b, h_Matrix* c, hipblasOperation_t op1, hipblasOperation_t op2, double* alpha){
  	 int m;
     int n;

	 if(op1 == HIPBLAS_OP_N && op2 == HIPBLAS_OP_N){
		 m = a->height;
		 n = b->width;
		 if( a->width != b->height ){
			 __threadfence();
			 asm("trap;");
		 }
	 }else if(op1 == HIPBLAS_OP_T && op2 == HIPBLAS_OP_N){
		 m = a->width;
		 n = b->width;
		 if( a->height != b->height ){
			 __threadfence();
			 asm("trap;");
		 }
	 }else if(op1 == HIPBLAS_OP_N && op2 == HIPBLAS_OP_T){
		 m = a->height;
		 n = b->height;
		 if( a->width != b->width ){
			 __threadfence();
			 asm("trap;");
		 }
	 }else{
		 m = a->width;
		 n = b->height;
		 if( a->height != b->width ){
			 __threadfence();
			 asm("trap;");
		 }
	 }

  	 int k = a->height;
	 if(op1 == HIPBLAS_OP_N){
	 	k = a->width;
	 }

	 int leadingDimensionA = a->height;
	 int leadingDimensionB = b->height;
	 int leadingDimensionC = a->height;

	 double* beta = new double[1]();
	 multiplyCuda(a->elements, b->elements, c->elements, leadingDimensionA, leadingDimensionB, leadingDimensionC, m, n, k, op1, op2, alpha, beta);
}

__global__ void matrixMultiplyCudaKernel(h_Matrix* a, h_Matrix* b, h_Matrix* c, int m, int n, int k, hipblasOperation_t op1, hipblasOperation_t op2, double* alpha){
	matrixMultiplyCuda(a, b, c, m, n, k, op1, op2, alpha);
}

__global__ void matrixMultiplyCudaKernel(h_Matrix* a, h_Matrix* b, h_Matrix* c, hipblasOperation_t op1, hipblasOperation_t op2, double* alpha){
	matrixMultiplyCuda(a, b, c, op1 ,op2, alpha);
}




h_Matrix* copyMatrixToDevice(h_Matrix *hostMatrix){
    h_Matrix *deviceMatrix;
    double *deviceElements;

    // Allocate Space on Device for Array
    cudaCheck(hipMalloc(&deviceElements, hostMatrix->numel() * sizeof(double)));
    hostMatrix->devElements = deviceElements;

    // Allocate Space on Device for Class Container
    cudaCheck(hipMalloc((void **)&deviceMatrix, sizeof(h_Matrix)));

    // Copy contents of host matrix to the device matrix container
    cudaCheck(hipMemcpy(deviceMatrix, &hostMatrix, sizeof(h_Matrix), hipMemcpyHostToDevice));

    // Copy Contents of Array from host to device
    cudaCheck(hipMemcpy(deviceElements, hostMatrix->elements, hostMatrix->numel() * sizeof(double), hipMemcpyHostToDevice));

    // Copy address of array to matrix elements
    cudaCheck(hipMemcpy(&(deviceMatrix->elements), &deviceElements, sizeof(double *), hipMemcpyHostToDevice));

    // Copy remaining members to device
    cudaCheck(hipMemcpy(&(deviceMatrix->height), &hostMatrix->height, sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(&(deviceMatrix->width), &hostMatrix->width, sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(&(deviceMatrix->depth), &hostMatrix->depth, sizeof(int), hipMemcpyHostToDevice));

    return deviceMatrix;
}

void copyMatrixToHost(h_Matrix *hostMatrix, h_Matrix *deviceMatrix){

    double *hostElements;

	// Copy Matrix Properties
    cudaCheck(hipMemcpy(&hostMatrix->height, &deviceMatrix->height, sizeof(int),hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(&hostMatrix->width, &deviceMatrix->width, sizeof(int),hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(&hostMatrix->depth, &deviceMatrix->depth, sizeof(int),hipMemcpyDeviceToHost));


    // Copy Elements from Device to Host)
    if(!hostMatrix->elements){
    	delete [] hostMatrix->elements;
    }

    hostElements = new double[hostMatrix->numel()];
    hostMatrix->elements = hostElements;

    // Copy the value (address) of elements on the device to the host devElements attribute
    cudaCheck(hipMemcpy(&hostMatrix->devElements, &deviceMatrix->elements, sizeof(double *), hipMemcpyDeviceToHost));

    // Copy the elements from the device to the elements container on the host
    cudaCheck(hipMemcpy(hostMatrix->elements, hostMatrix->devElements,sizeof(double) * hostMatrix->numel(),hipMemcpyDeviceToHost));

}

h_Matrix copyMatrixToHost(h_Matrix *deviceMatrix){
    //Allocate Space on Host for Class Container)
	h_Matrix hostMatrix;
    copyMatrixToHost(&hostMatrix, deviceMatrix);
    return hostMatrix;
}




__global__ void d_IPSP3d(h_Matrix* re, h_Matrix* v1, h_Matrix* v2, h_Matrix* v3, h_Matrix* cc){

	double scalar = 1;
	__shared__ int n1, l3;)
    __shared__ h_Matrix aMatrix;
	__shared__ double *aMatrixElements;

	if(threadIdx.x == 0 && threadIdx.y == 0){
		n1 = v1->width;
		l3 = v3->height;
		aMatrixElements = new double[re->width]();
		aMatrix.elements = aMatrixElements;
		aMatrix.width =; 8; aMatrix.height = aMatrix.depth = 1;
	}

		for(int i = 0; i < n1; i++){
			cc->setElement(i, 0.0);
			for(int j = 0; j < l3; j++){
				h_Matrix v1Col = v1->getCol(i);
				h_Matrix v2Col = v2->getCol(i);
				matrixMultiplyCuda(&v1Col, &re->getPlane(j), &aMatrix, 1, re->width, v1->height, HIPBLAS_OP_T, HIPBLAS_OP_N, &scalar);
				__syncthreads();
				matrixMultiplyCuda(&aMatrix, &v2Col, cc->getElement(i), 1, 1, aMatrix.width, HIPBLAS_OP_N, HIPBLAS_OP_N, v3->getElement(i, j), &scalar);
				__syncthreads();
				//if (threadIdx.x == 0){printf("i: %d, j: %d, V3 Val:%f, \n", i, j, v3->getElement(i,j)[0]);}
			}
		}
    return;
}


int main() {


    double dxElements[] = {0.3536, 0.3536, 0.3536, 0.3536, 0.3536, 0.3536, 0.3536, 0.3536, 0.4976, 0.4785, 0.4410, 0.3865, 0.3172, 0.2357, 0.1451, 0.0490, 0.4904, 0.4157, 0.2778, 0.0975, -0.0975, -0.2778, -0.4157, -0.4904, 0.4785, 0.3172, 0.0490, -0.2357, -0.4410, -0.4976, -0.3865, -0.1451, 0.4619, 0.1913, -0.1913, -0.4619, -0.4619, -0.1913, 0.1913, 0.4619, 0.4410, 0.0490, -0.3865, -0.4785, -0.1451, 0.3172, 0.4976, 0.2357, 0.4157, -0.0975, -0.4904, -0.2778, 0.2778, 0.4904, 0.0975, -0.4157, 0.3865, -0.2357, -0.4785, 0.0490, 0.4976, 0.1451, -0.4410, -0.3172, 0.3536, -0.3536, -0.3536, 0.3536, 0.3536, -0.3536, -0.3536, 0.3536, 0.3172, -0.4410, -0.1451, 0.4976, -0.0490, -0.4785, 0.2357, 0.3865, 0.2778, -0.4904, 0.0975, 0.4157, -0.4157, -0.0975, 0.4904, -0.2778, 0.2357, -0.4976, 0.3172, 0.1451, -0.4785, 0.3865, 0.0490, -0.4410, 0.1913, -0.4619, 0.4619, -0.1913, -0.1913, 0.4619, -0.4619, 0.1913, 0.1451, -0.3865, 0.4976, -0.4410, 0.2357, 0.0490, -0.3172, 0.4785, 0.0975, -0.2778, 0.4157, -0.4904, 0.4904, -0.4157, 0.2778, -0.0975, 0.0490, -0.1451, 0.2357, -0.3172, 0.3865, -0.4410, 0.4785, -0.4976, 0.0490, 0.1451, 0.2357, 0.3172, 0.3865, 0.4410, 0.4785, 0.4976, 0.0975, 0.2778, 0.4157, 0.4904, 0.4904, 0.4157, 0.2778, 0.0975, 0.1451, 0.3865, 0.4976, 0.4410, 0.2357, -0.0490, -0.3172, -0.4785, 0.1913, 0.4619, 0.4619, 0.1913, -0.1913, -0.4619, -0.4619, -0.1913, 0.2357, 0.4976, 0.3172, -0.1451, -0.4785, -0.3865, 0.0490, 0.4410, 0.2778, 0.4904, 0.0975, -0.4157, -0.4157, 0.0975, 0.4904, 0.2778, 0.3172, 0.4410, -0.1451, -0.4976, -0.0490, 0.4785, 0.2357, -0.3865, 0.3536, 0.3536, -0.3536, -0.3536, 0.3536, 0.3536, -0.3536, -0.3536, 0.3865, 0.2357, -0.4785, -0.0490, 0.4976, -0.1451, -0.4410, 0.3172, 0.4157, 0.0975, -0.4904, 0.2778, 0.2778, -0.4904, 0.0975, 0.4157, 0.4410, -0.0490, -0.3865, 0.4785, -0.1451, -0.3172, 0.4976, -0.2357, 0.4619, -0.1913, -0.1913, 0.4619, -0.4619, 0.1913, 0.1913, -0.4619, 0.4785, -0.3172, 0.0490, 0.2357, -0.4410, 0.4976, -0.3865, 0.1451, 0.4904, -0.4157, 0.2778, -0.0975, -0.0975, 0.2778, -0.4157, 0.4904, 0.4976, -0.4785, 0.4410, -0.3865, 0.3172, -0.2357, 0.1451, -0.0490, 0.3536, -0.3536, 0.3536, -0.3536, 0.3536, -0.3536, 0.3536, -0.3536, 1.0000, 0, 0, 0, 0, 0, 0, 0, 0, 1.0000, 0, 0, 0, 0, 0, 0, 0, 0, 1.0000, 0, 0, 0, 0, 0, 0, 0, 0, 1.0000, 0, 0, 0, 0, 0, 0, 0, 0, 1.0000, 0, 0, 0, 0, 0, 0, 0, 0, 1.0000, 0, 0, 0, 0, 0, 0, 0, 0, 1.0000, 0, 0, 0, 0, 0, 0, 0, 0, 1.0000};
    double dyElements[] = {0.3536, 0.3536, 0.3536, 0.3536, 0.3536, 0.3536, 0.3536, 0.3536, 0.4976, 0.4785, 0.4410, 0.3865, 0.3172, 0.2357, 0.1451, 0.0490, 0.4904, 0.4157, 0.2778, 0.0975, -0.0975, -0.2778, -0.4157, -0.4904, 0.4785, 0.3172, 0.0490, -0.2357, -0.4410, -0.4976, -0.3865, -0.1451, 0.4619, 0.1913, -0.1913, -0.4619, -0.4619, -0.1913, 0.1913, 0.4619, 0.4410, 0.0490, -0.3865, -0.4785, -0.1451, 0.3172, 0.4976, 0.2357, 0.4157, -0.0975, -0.4904, -0.2778, 0.2778, 0.4904, 0.0975, -0.4157, 0.3865, -0.2357, -0.4785, 0.0490, 0.4976, 0.1451, -0.4410, -0.3172, 0.3536, -0.3536, -0.3536, 0.3536, 0.3536, -0.3536, -0.3536, 0.3536, 0.3172, -0.4410, -0.1451, 0.4976, -0.0490, -0.4785, 0.2357, 0.3865, 0.2778, -0.4904, 0.0975, 0.4157, -0.4157, -0.0975, 0.4904, -0.2778, 0.2357, -0.4976, 0.3172, 0.1451, -0.4785, 0.3865, 0.0490, -0.4410, 0.1913, -0.4619, 0.4619, -0.1913, -0.1913, 0.4619, -0.4619, 0.1913, 0.1451, -0.3865, 0.4976, -0.4410, 0.2357, 0.0490, -0.3172, 0.4785, 0.0975, -0.2778, 0.4157, -0.4904, 0.4904, -0.4157, 0.2778, -0.0975, 0.0490, -0.1451, 0.2357, -0.3172, 0.3865, -0.4410, 0.4785, -0.4976, 0.0490, 0.1451, 0.2357, 0.3172, 0.3865, 0.4410, 0.4785, 0.4976, 0.0975, 0.2778, 0.4157, 0.4904, 0.4904, 0.4157, 0.2778, 0.0975, 0.1451, 0.3865, 0.4976, 0.4410, 0.2357, -0.0490, -0.3172, -0.4785, 0.1913, 0.4619, 0.4619, 0.1913, -0.1913, -0.4619, -0.4619, -0.1913, 0.2357, 0.4976, 0.3172, -0.1451, -0.4785, -0.3865, 0.0490, 0.4410, 0.2778, 0.4904, 0.0975, -0.4157, -0.4157, 0.0975, 0.4904, 0.2778, 0.3172, 0.4410, -0.1451, -0.4976, -0.0490, 0.4785, 0.2357, -0.3865, 0.3536, 0.3536, -0.3536, -0.3536, 0.3536, 0.3536, -0.3536, -0.3536, 0.3865, 0.2357, -0.4785, -0.0490, 0.4976, -0.1451, -0.4410, 0.3172, 0.4157, 0.0975, -0.4904, 0.2778, 0.2778, -0.4904, 0.0975, 0.4157, 0.4410, -0.0490, -0.3865, 0.4785, -0.1451, -0.3172, 0.4976, -0.2357, 0.4619, -0.1913, -0.1913, 0.4619, -0.4619, 0.1913, 0.1913, -0.4619, 0.4785, -0.3172, 0.0490, 0.2357, -0.4410, 0.4976, -0.3865, 0.1451, 0.4904, -0.4157, 0.2778, -0.0975, -0.0975, 0.2778, -0.4157, 0.4904, 0.4976, -0.4785, 0.4410, -0.3865, 0.3172, -0.2357, 0.1451, -0.0490, 0.3536, -0.3536, 0.3536, -0.3536, 0.3536, -0.3536, 0.3536, -0.3536, 1.0000, 0, 0, 0, 0, 0, 0, 0, 0, 1.0000, 0, 0, 0, 0, 0, 0, 0, 0, 1.0000, 0, 0, 0, 0, 0, 0, 0, 0, 1.0000, 0, 0, 0, 0, 0, 0, 0, 0, 1.0000, 0, 0, 0, 0, 0, 0, 0, 0, 1.0000, 0, 0, 0, 0, 0, 0, 0, 0, 1.0000, 0, 0, 0, 0, 0, 0, 0, 0, 1.0000};
    double dzElements[] = {0.5774,0.5774,0.5774,0.7887,0.5774,0.2113,0.7071,0.0000,-0.7071,0.5774,-0.5774,-0.5774,0.4082,-0.8165,0.4082,0.2113,-0.5774,0.7887,0.2113,0.5774,0.7887,0.4082,0.8165,0.4082,0.5774,0.5774,-0.5774,0.7071,0.0000,-0.7071,0.7887,-0.5774,0.2113,0.5774,-0.5774,0.5774,1.0000,0,0,0,1.0000,0,0,0,1.0000};
    double reElements[] = {8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,8,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,7,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6,6};
    double* ccElements = new double[40]();

    printDeviceDetails();

	//Initialise Matrix containers
    std::cout << "Initializing.." << std::endl;
    h_Matrix h_dx(dxElements, 8, 1, 1);
    h_Matrix h_dy(dyElements, 8, 1, 1);
    h_Matrix h_dz(dzElements, 3, 1, 1);
    h_Matrix h_re(reElements, 8, 8, 8);
    h_Matrix h_cc(ccElements, 1, 40, 1);

    hipDeviceSetLimit(hipLimitMallocHeapSize, 1024*1024*32);

    //Copy Matrix's to    std::cout << "Device Details... \n" << std::endl;
    std::cout << "Copying to Device.." << std::endl;
    h_Matrix* d_dx = copyMatrixToDevice(&h_dx);
    h_Matrix* d_dy = copyMatrixToDevice(&h_dy);
    h_Matrix* d_dz = copyMatrixToDevice(&h_dz);
    h_Matrix* d_re = copyMatrixToDevice(&h_re);
    h_Matrix* d_cc = copyMatrixToDevice(&h_cc);

    std::cout << "Starting.." <<std::endl;

    double* cElements = new double[h_re.width]();
    h_Matrix h_c(cElements, 1, 8, 1);
    h_Matrix* d_c = copyMatrixToDevice(&h_c);

    double *devScalar;
    double scalar = 1;

    cudaCheck( hipMalloc(&devScalar, sizeof(double)));
    cudaCheck( hipMemcpy(devScalar, &scalar, sizeof(double), hipMemcpyHostToDevice));


    //status = hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 1, h_dx.height, h_re.height, &scalar, &d_dx->elements[0], h_dx.height, &d_re->elements[0], h_re.height, 0, &aMatrix[0], 1);
    //status = hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, h_re.height, h_re.height, 1, &scalar, &d_dx->elements[0], 1, &d_re->elements[0], 1, 0, &aMatrix[0], h_re.height);
    //MatrixMultiplyBLAS(handle, d_dx->elements, d_re->elements, aMatrix, h_dx.height, 1, h_re.width, HIPBLAS_OP_T, HIPBLAS_OP_N);
    //matrixMultiplyCuda<<<1,threadsPerBlock>>>(d_dx, d_re, d_c, HIPBLAS_OP_T, HIPBLAS_OP_N, devScalar);

    dim3 threadsPerBlock(8, 8);
    for(int i = 0; i < 1000; i++){
    d_IPSP3d<<< 1, threadsPerBlock>>>(d_re, d_dx, d_dy, d_dz, d_cc);
    }


    //matrixMultiplyCudaKernel<<<1,threadsPerBlock>>>(d_dx, d_re, d_c, 1, h_re.width, h_dx.height, HIPBLAS_OP_T, HIPBLAS_OP_N, devScalar);
   // double* testOutput = new double[h_dx.height * h_re.width]();

    // Either works :)
    //h_Matrix results = 	copyMatrixToHost(d_cc);
    copyMatrixToHost(&h_cc, d_cc);

    printf("%d,%d,%d, %f\n", h_cc.height, h_cc.width, h_cc.depth, h_cc.elements[0]);

    for(int i = 0; i < h_cc.height * h_cc.width * h_cc.depth; i++){
    	std::cout << h_cc.elements[i] << ", " << std::endl;
    }

    hipError_t err = hipGetLastError();
    
    printf("\nError: %s\n", hipGetErrorString(err));

    hipFree(d_dx);
    hipFree(d_dy);
    hipFree(d_dz);
    hipFree(d_re);
    hipFree(d_cc);
    return 0;
}

#endif

