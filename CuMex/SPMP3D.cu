#include "hip/hip_runtime.h"
#ifndef SPMP3D
#define SPMP3D

#include "h_Matrix.h"
#include "loadImage.cpp"
#include "CreateDict.cpp"
#include "Routines.cu"

#ifndef CUDACHECK
#define CUDACHECK
#define cudaCheck(input){cudaAssert((input), __FILE__, __LINE__); } // http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
inline void cudaAssert(hipError_t code, const char *file, int line){
	if (code != hipSuccess){ fprintf(stderr, "CudaAssert %s %s %d\n", hipGetErrorString(code), file, line); exit(code);}
}
#endif

void startSPMP3DRoutine(int blockSize, h_Matrix* inputImage, h_Matrix* Dx, h_Matrix* Dy, h_Matrix* Dz){
	int numberOfConcurrentBlocks = getNumberOfSMPs();

	dim3 threadsPerBlock(blockSize, blockSize,3);
	int nBlocks = (inputImage->height/8) * (inputImage->width/8);
	dim3 numberOfBlocks(inputImage->height/8, inputImage->width/8);
	//dim3 numberOfBlocks(3, 1);
	std::cout << "Initializing.." << std::endl;
	h_Matrix h_h((inputImage->height/8)*8, (inputImage->width/8)*8, (inputImage->depth/3)*3);

	double* h_c = new double[(inputImage->height/8)*8 * (inputImage->width/8)*8 *  (inputImage->depth/3)*3 * 2]();
//	h_Matrix h_c((inputImage->height/8)*8, (inputImage->width/8)*8, (inputImage->depth/3)*3);

	cudaCheck( hipDeviceSetLimit(hipLimitMallocHeapSize, 1024*1024*256) );

	//Copy Matrix'sglobal to device
	std::cout << "Copying to Device.." << std::endl;
	//printf("%d, %d, %d\n", inputImage->height, inputImage->width, inputImage->depth);
	h_Matrix* d_f = copyMatrixToDevice(inputImage);
	h_Matrix* d_dx = copyMatrixToDevice(Dx);
	h_Matrix* d_dy = copyMatrixToDevice(Dy);
	h_Matrix* d_dz = copyMatrixToDevice(Dz);
//	h_Matrix* d_c = copyMatrixToDevice(&h_c);
	h_Matrix* d_h = copyMatrixToDevice(&h_h);

	//h_h.height = 8; h_h.width = 8; h_h.depth = 3;
	//h_c.height = 8; h_c.width = 8; h_c.depth = 3;
	double pss = 40.5;
	// Declare host variables
	double h_tol = pow(255,2)/(pow(10,pss/10.0));//5.7954;
	double h_No = inputImage->numel()/15.0;//1.3107e+4;
	double h_toln = 1e-8;
	int h_lstep = -1;
	int h_Max = 50000;
	//int h_Max = 5;
	int h_Maxp = 50000;
	//int h_Maxp = 5;
	int* h_Set_ind = new int[8*8*3]();
	int* h_numat = new int[nBlocks]();
	double* d_c;

	// Declare device variables
	double* d_tol;
	double* d_No;
	double* d_toln;
	int* d_lstep;
	int* d_Max;
	int* d_Maxp ;
	int* d_Set_ind;
	int* d_numat;

	// Allocate Space on device
	cudaCheck( hipMalloc( &d_tol, sizeof(double) ) );
	cudaCheck( hipMalloc( &d_No, sizeof(double) ) );
	cudaCheck( hipMalloc( &d_toln, sizeof(double) ) );
	cudaCheck( hipMalloc( &d_lstep, sizeof(int) ) );
	cudaCheck( hipMalloc( &d_Max, sizeof(int) ) );
	cudaCheck( hipMalloc( &d_Maxp, sizeof(int) ) );
	cudaCheck( hipMalloc( &d_Set_ind, sizeof(int) * Dx->height * Dy->height * Dz->height ) );
	cudaCheck( hipMalloc( &d_numat, sizeof(int) * nBlocks) );
	cudaCheck( hipMalloc( &d_c, sizeof(double) * (inputImage->height/8)*8 * (inputImage->width/8)*8 *  (inputImage->depth/3)*3 * 2) );

	// Initalise values on device
	cudaCheck( hipMemcpy( d_tol, &h_tol, sizeof(double), hipMemcpyHostToDevice));
	cudaCheck( hipMemcpy( d_No, &h_No, sizeof(double), hipMemcpyHostToDevice));
	cudaCheck( hipMemcpy( d_toln, &h_toln, sizeof(double), hipMemcpyHostToDevice));
	cudaCheck( hipMemcpy( d_lstep, &h_lstep, sizeof(int), hipMemcpyHostToDevice));
	cudaCheck( hipMemcpy( d_Max, &h_Max, sizeof(int), hipMemcpyHostToDevice));
	cudaCheck( hipMemcpy( d_Maxp, &h_Maxp, sizeof(int), hipMemcpyHostToDevice));
	cudaCheck( hipMemcpy( d_Maxp, &h_Maxp, sizeof(int), hipMemcpyHostToDevice));
	cudaCheck( hipMemcpy( d_Set_ind, h_Set_ind, sizeof(int) * 192, hipMemcpyHostToDevice));
	cudaCheck( hipMemcpy( d_numat, h_numat, sizeof(int) * nBlocks, hipMemcpyHostToDevice));
	cudaCheck( hipMemcpy( d_c, h_c, sizeof(double) * (inputImage->height/8)*8 * (inputImage->width/8)*8 *  (inputImage->depth/3)*3 * 2, hipMemcpyHostToDevice));

	int blockDim = Dx->height * Dy->height * Dz->height;
	printf("Tolerance: %f, Max Iterations %d, PSS: %f, LStep: %d, h_No: %f\nXBlocks: %d, YBlocks:%d\n", h_tol, h_Max, pss, h_lstep, h_No,(inputImage->width/8),(inputImage->height/8));
	std::cout << "Calculating..";
	// *4 for extra c
	d_SPMP3DKernel<<< numberOfBlocks, threadsPerBlock,(blockDim * 4 + Dx->numel() + Dy->numel() + Dz->numel() + Dx->numel()) * sizeof(double) >>>(d_f, d_dx, d_dy, d_dz, d_tol, d_No, d_toln, d_lstep, d_Max, d_Maxp, d_h, d_c, d_Set_ind, d_numat);
	//d_SPMP3DKernel<<< 14, threadsPerBlock, (blockDim + Dx->numel() + Dy->numel() + Dz->numel() + Dx->numel() + h_h.numel() + h_c.numel()) * sizeof(double)>>>(d_f, d_dx, d_dy, d_dz, d_tol, d_No, d_toln, d_lstep, d_Max, d_Maxp, d_h, d_c, d_Set_ind, d_numat);
	cudaCheck( hipDeviceSynchronize() );
	std::cout << "Done." << std::endl;
	std::cout << "Outcome.." << std::endl;
	hipError_t err = hipGetLastError();	
	std::cout << err << std::endl;
	
	cudaCheck( hipMemcpy( h_numat, d_numat, sizeof(int) * nBlocks, hipMemcpyDeviceToHost))
	int numberOfCoeffs = 0;
	for(int i = 0; i < nBlocks; i++){
		numberOfCoeffs += h_numat[i];
	}
	std::cout << "numberOfCoeffs.. " << numberOfCoeffs << std::endl;
	std::cout << "Sparsity Ratio.. " << inputImage->numel() / (double)numberOfCoeffs << std::endl;
//	std::cout << numberOfCoeffs << std::endl;
	delete [] h_Set_ind;

	cudaCheck( hipFree(d_tol) );
	cudaCheck( hipFree(d_No) );
	cudaCheck( hipFree(d_toln) );
	cudaCheck( hipFree(d_lstep) );
	cudaCheck( hipFree(d_Max) );
	cudaCheck( hipFree(d_Maxp) );
	cudaCheck( hipFree(d_Set_ind) );
	cudaCheck( hipFree(d_numat) );

	cudaCheck( hipFree(d_f) );
	cudaCheck( hipFree(d_dx) );
	cudaCheck( hipFree(d_dy) );
	cudaCheck( hipFree(d_dz) );
	cudaCheck( hipFree(d_c) );
	cudaCheck( hipFree(d_h) );


}

int main(int argc, char** argv )
{
    if ( argc != 2 ) { printf("usage: SPMP3D.out <Image_Path>\n"); return -1; }

    cudaCheck( hipDeviceReset() );

    printDeviceDetails();

    //cudaCheck( hipDeviceSetLimit(hipLimitMallocHeapSize, 1024*1024*32) );
    cudaCheck( hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte) );
    //hipFuncSetCacheConfig(reinterpret_cast<const void*>(d_SPMP3D), inputImagecudaFuncCachePreferShared);
    cudaCheck( hipDeviceSetCacheConfig(hipFuncCachePreferShared) );

    h_Matrix* inputImage = loadImageToMatrix(argv[1]);
 //   h_Matrix* testImage = new h_Matrix(inputImage->elements[8*8], inputImage->height, inputImage->width, inputImage->depth);

    h_Matrix Dx = createStandardDict();
    h_Matrix Dy = createStandardDict();
    h_Matrix Dz = createDzDict();

    startSPMP3DRoutine(8, inputImage, &Dx, &Dy, &Dz);

    //for(int i = 0; i < Dx.numel(); i ++){ printf("%f, ", Dx.elements[i]); }
    //std::cout << "\n\n";
    //for(int i = 0; i < Dz.numel(); i ++){ printf("%f, ", Dz.elements[i]); }
    //waitKey(0);
    delete inputImage;
    return 0;
}




#endif

